#include "hip/hip_runtime.h"
#include "CommonFunctions.cuh"

#include "math.h"
#include ""
#include "device_atomic_functions.h"
//#include "device_types.h"

#include "Platform.cuh"
#include "vmath.hpp"

#include "BruteforceVariables.cuh"
#include "Floors.cuh"
#include "RunParameters.hpp"


void write_run_parameters(std::ofstream& wfrp, std::string timestamp)
{
    wfrp << std::fixed;

    wfrp << "Run Timestamp: " << timestamp << "\n\n";

    wfrp << "nThreads: " << nThreads << '\n';
    wfrp << "memorySize: " << memorySize << "\n\n";

    if (computeMaxElevation)
        wfrp << "Computing Max Elevation!\n\n";

    wfrp << "Solver Mode: " << solverMode << "\n\n";

    wfrp << "Is ZXSum: " << useZXSum << "\n\n";

    if (useZXSum)
        wfrp << "Use Positive Z: " << usePositiveZ << "\n\n";

    wfrp << "MinQ1: " << minQ1 << '\n';
    wfrp << "MaxQ1: " << maxQ1 << '\n';
    wfrp << "MinQ2: " << minQ2 << '\n';
    wfrp << "MaxQ2: " << maxQ2 << '\n';
    wfrp << "MinQ3: " << minQ3 << '\n';
    wfrp << "MaxQ3: " << maxQ3 << "\n\n";

    wfrp << "nPUFrames: " << nPUFrames << '\n';
    wfrp << "maxFrames: " << maxFrames << "\n\n";

    wfrp << "minNX: " << minNX << '\n';
    wfrp << "maxNX: " << maxNX << '\n';
    wfrp << "minNY: " << minNY << '\n';
    wfrp << "maxNY: " << maxNY << '\n';

    if (!useZXSum)
    {
        wfrp << "minNZ: " << minNZ << '\n';
        wfrp << "maxNZ: " << maxNZ << "\n\n";
    }
    else
    {
        wfrp << "minNZXSum: " << minNZXSum << '\n';
        wfrp << "maxNZXSum: " << maxNZXSum << "\n\n";
    }

    wfrp << "nSamplesNX: " << nSamplesNX << '\n';
    wfrp << "nSamplesNZ: " << nSamplesNZ << '\n';
    wfrp << "nSamplesNY: " << nSamplesNY << "\n\n";

    wfrp << "deltaX: " << deltaX << '\n';
    wfrp << "deltaZ: " << deltaZ << "\n\n";

    wfrp << "NormalListPath: " << normalsInput << "\n\n";

    wfrp << "Platform Position: " << platformPos[0] << ", " << platformPos[1] << ", " << platformPos[2] << "\n";
}

void print_help(std::string &default_output_path, std::string &default_run_params_path)
{
    printf("BitFS Platform Max Tilt Brute Forcer.\n");
    printf("This program accepts the following options:\n\n");
    printf("-f <frames>: Maximum frames of platform tilt considered.\n");
    printf("             Default: %d\n", maxFrames);
    printf("-p <frames>: Number of frames of PU movement for 10k glitch\n");
    printf("             Default: %d\n", nPUFrames);
    printf("-q1 <min_q1> <max_q1>: Range of q-frames to test for frame 1 of 10k PU route.\n");
    printf("                       Default: %d %d\n", minQ1, maxQ1);
    printf("-q2 <min_q2> <max_q2>: Range of q-frames to test for frame 2 of 10k PU route.\n");
    printf("                       Default: %d %d\n", minQ2, maxQ2);
    printf("-q3 <min_q3> <max_q3>: Range of q-frames to test for frame 3 of 10k PU route.\n");
    printf("                       Default: %d %d\n", minQ3, maxQ3);
    printf("-nx <min_nx> <max_nx> <n_samples>: Inclusive range of x normals to be considered, and the number of normals to sample.\n");
    printf("                                   If min_nx==max_nx then n_samples will be set to 1.\n");
    printf("                                   If a list of normals is provided, then these parameters will define displacements from each normal.\n");
    printf("                                   Default: %g %g %d\n", minNX, maxNX, nSamplesNX);
    printf("-nz <min_nz> <max_nz> <n_samples>: Inclusive range of z normals to be considered, and the number of normals to sample.\n");
    printf("                                   ONLY USED IF -sum IS SET TO 0.\n");
    printf("                                   If min_nz==max_nz then n_samples will be set to 1.\n");
    printf("                                   If a list of normals is provided, then these parameters will define displacements from each normal.\n");
    printf("                                   Default: %g %g %d\n", minNZ, maxNZ, nSamplesNZ);
    printf("-nzxsum <min_nzxsum> <max_nzxsum> <n_samples>: Inclusive range of zxsum normals to be considered, and the number of normals to sample.\n");
    printf("                                               ONLY USED IF -sum IS SET TO 1.\n");
    printf("                                               If min_nz==max_nz then n_samples will be set to 1.\n");
    printf("                                               If a list of normals is provided, then these parameters will define displacements from each normal.\n");
    printf("                                               Default: %g %g %d\n", minNZ, maxNZ, nSamplesNZ);
    printf("-ny <min_ny> <max_ny> <n_samples>: Inclusive range of y normals to be considered, and the number of normals to sample.\n");
    printf("                                   If min_ny==max_ny then n_samples will be set to 1.\n");
    printf("                                   If a list of normals is provided, then these parameters will define displacements from each normal.\n");
    printf("                                   Default: %g %g %d\n", minNY, maxNY, nSamplesNY);
    printf("-dx <delta_x>: x coordinate spacing of positions on the platform.\n");
    printf("               Default: %g\n", deltaX);
    printf("-dz <delta_z>: z coordinate spacing of positions on the platform.\n");
    printf("               Default: %g\n", deltaZ);
    printf("-p <platform_x> <platform_y> <platform_z>: Position of the pyramid platform.\n");
    printf("                                           Default: %g %g %g\n", platformPos[0], platformPos[1], platformPos[2]);
    printf("-solver <0, 1, or 2>: Flag for which solver to use (0 for non-HAU-Aligned, 1 for HAU-Aligned, 2 for Slide Kick).\n");
    printf("                      Default: %i\n", solverMode);
    printf("-ni: Optional path to a list of normals around which to sample. If left empty, no list of normals is used, and samples are displaced from (0,0,0).\n");
    printf("    Default: %s\n", normalsInput.c_str());
    printf("-o: Path to the output file.\n");
    printf("    Default: %s\n", default_output_path.c_str());
    printf("-rp: Path to the run parameters file.\n");
    printf("     Default: %s\n", default_run_params_path.c_str());
    printf("-sum <0 or 1>: Flag for whether to parameterize by Z or by ZXSum (0 for Z, 1 for ZXSum).\n");
    printf("               Default: %i\n", useZXSum);
    printf("-posZ <0 or 1>: Flag for whether to use postive Z or negative Z (0 for -Z, 1 for +Z).\n");
    printf("                Only used when parameterizing by ZXSum instead of Z.\n");
    printf("                Default: %i\n", usePositiveZ);
    printf("-ssp <0, 1, or 2>: Printing mode for subsolutions (0 for no subsolution printing, 1 for minimal printing, 2 for full printing).\n");
    printf("                   Default: %i\n", subSolutionPrintingMode);
    printf("-t <threads>: Number of CUDA threads to assign to the program.\n");
    printf("              Default: %d\n", nThreads);
    printf("-m <memory>: Amount of GPU memory to assign to the program.\n");
    printf("             Default: %d\n", memorySize);
    printf("-v: Verbose mode. Prints all parameters used in brute force.\n");
    printf("    Default: off\n");
    printf("-h --help: Prints this text.\n");
    exit(0);
}

void process_argument(int& i, char* argv[], std::string& outFileSolutionData, std::string& outFileRunParams)
{
    if (!strcmp(argv[i], "-h") || !strcmp(argv[i], "--help")) {
        print_help(outFileSolutionData, outFileRunParams);
    }
    else if (!strcmp(argv[i], "-f")) {
        maxFrames = std::stoi(argv[i + 1]);

        i += 1;
    }
    else if (!strcmp(argv[i], "-q1")) {
        minQ1 = std::stoi(argv[i + 1]);
        maxQ1 = std::stoi(argv[i + 2]);

        i += 2;
    }
    else if (!strcmp(argv[i], "-q2")) {
        minQ2 = std::stoi(argv[i + 1]);
        maxQ2 = std::stoi(argv[i + 2]);

        i += 2;
    }
    else if (!strcmp(argv[i], "-q3")) {
        minQ3 = std::stoi(argv[i + 1]);
        maxQ3 = std::stoi(argv[i + 2]);

        i += 2;
    }
    else if (!strcmp(argv[i], "-p")) {
        nPUFrames = std::stoi(argv[i + 1]);

        i += 1;
    }
    else if (!strcmp(argv[i], "-t")) {
        nThreads = std::stoi(argv[i + 1]);

        i += 1;
    }
    else if (!strcmp(argv[i], "-m")) {
        memorySize = std::stoi(argv[i + 1]);

        i += 1;
    }
    else if (!strcmp(argv[i], "-nx")) {
        minNX = std::stof(argv[i + 1]);
        maxNX = std::stof(argv[i + 2]);

        if (minNX == maxNX) {
            nSamplesNX = 1;
        }
        else {
            nSamplesNX = std::stoi(argv[i + 3]);
        }

        i += 3;
    }
    else if (!strcmp(argv[i], "-nz")) {
        minNZ = std::stof(argv[i + 1]);
        maxNZ = std::stof(argv[i + 2]);

        if (minNZ == maxNZ) {
            nSamplesNZ = 1;
        }
        else {
            nSamplesNZ = std::stoi(argv[i + 3]);
        }

        i += 3;
    }
    else if (!strcmp(argv[i], "-nzxsum")) {
        minNZXSum = std::stof(argv[i + 1]);
        maxNZXSum = std::stof(argv[i + 2]);

        if (minNZXSum == maxNZXSum) {
            nSamplesNZ = 1;
        }
        else {
            nSamplesNZ = std::stoi(argv[i + 3]);
        }

        i += 3;
    }
    else if (!strcmp(argv[i], "-ny")) {
        minNY = std::stof(argv[i + 1]);
        maxNY = std::stof(argv[i + 2]);

        if (minNY == maxNY) {
            nSamplesNY = 1;
        }
        else {
            nSamplesNY = std::stoi(argv[i + 3]);
        }

        i += 3;
    }
    else if (!strcmp(argv[i], "-dx")) {
        deltaX = std::stof(argv[i + 1]);
        i += 1;
    }
    else if (!strcmp(argv[i], "-dz")) {
        deltaZ = std::stof(argv[i + 1]);
        i += 1;
    }
    else if (!strcmp(argv[i], "-p")) {
        platformPos[0] = std::stof(argv[i + 1]);
        platformPos[1] = std::stof(argv[i + 2]);
        platformPos[2] = std::stof(argv[i + 3]);
        i += 3;
    }
    else if (!strcmp(argv[i], "-solver")) {
        solverMode = std::stoi(argv[i + 1]);
        i += 1;
    }
    else if (!strcmp(argv[i], "-ni")) {
        normalsInput = argv[i + 1];
        i += 1;
    }
    else if (!strcmp(argv[i], "-o")) {
        outFileSolutionData = argv[i + 1];
        i += 1;
    }
    else if (!strcmp(argv[i], "-rp")) {
        outFileRunParams = argv[i + 1];
        i += 1;
    }
    else if (!strcmp(argv[i], "-sum")) {
        useZXSum = std::stoi(argv[i + 1]);
        i += 1;
    }
    else if (!strcmp(argv[i], "-posZ")) {
        usePositiveZ = std::stoi(argv[i + 1]);
        i += 1;
    }
    else if (!strcmp(argv[i], "-ssp")) {
        subSolutionPrintingMode = std::stoi(argv[i + 1]);
        i += 1;
    }
    else if (!strcmp(argv[i], "-v")) {
        verbose = true;
    }
}

__global__ void print_success() {
    printf("CUDA code completed successfully.\n");
}

// This was taken from the following StackOverflow post: https://stackoverflow.com/a/51549250
__device__ float atomicMinFloat(float* addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value))) :
        __uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

    return old;
}

__device__ bool check_inbounds(const float* mario_pos) {
    short x_mod = (short)(int)mario_pos[0];
    short y_mod = (short)(int)mario_pos[1];
    short z_mod = (short)(int)mario_pos[2];

    return (abs(x_mod) < 8192 & abs(y_mod) < 8192 & abs(z_mod) < 8192);
}

__global__ void set_squish_ceilings(float n0, float n1, float n2, float n3) {
    squishCeilings[0] = n0 > -0.5;
    squishCeilings[1] = n1 > -0.5;
    squishCeilings[2] = n2 > -0.5;
    squishCeilings[3] = n3 > -0.5;
}

__global__ void set_platform_pos(float x, float y, float z) {
    platform_pos[0] = x;
    platform_pos[1] = y;
    platform_pos[2] = z;
}

__global__ void set_platform_normal(float nx, float ny, float nz) {
    platformNormal[0] = nx;
    platformNormal[1] = ny;
    platformNormal[2] = nz;
}

__global__ void calculate_10k_multipliers(int minQ1Q2, int maxQ1Q2, int minQ3, int maxQ3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < 112) {
        int temp = idx;
        int q1q2 = (temp % 7) + 2;
        temp = temp / 7;
        int q3 = (temp % 4) + 1;
        temp = temp / 4;
        int f = temp % 2;
        temp = temp / 2;
        int o = temp;

        if (q1q2 >= minQ1Q2 && q1q2 <= maxQ1Q2 && q3 >= minQ3 && q3 <= maxQ3) {
            tenKMultipliers[idx] = -(startNormals[f][1] + (double)q1q2 - 1.0) / ((o == 0 ? oneUpPlatformNormalYRight : oneUpPlatformNormalYLeft) + (double)q3 - 1.0);
        }
        else {
            tenKMultipliers[idx] = NAN;
        }
    }
}

__global__ void init_reverse_atan() {

    for (int i = 0; i < 8192; i++) {
        int angle = (65536 + gArctanTableG[i]) % 65536;
        gReverseArctanTable[angle] = i;
    }
}

__global__ void set_start_triangle(short* tris, float* norms) {
    for (int x = 0; x < 2; x++) {
        for (int y = 0; y < 3; y++) {
            startTriangles[x][y][0] = tris[9 * x + 3 * y];
            startTriangles[x][y][1] = tris[9 * x + 3 * y + 1];
            startTriangles[x][y][2] = tris[9 * x + 3 * y + 2];
            startNormals[x][y] = norms[3 * x + y];
        }
    }
}

__device__ int16_t atan2_lookupG(float z, float x) {
    int16_t angle = 0;

    if (x == 0) {
        angle = gArctanTableG[0];
    }
    else {
        angle = gArctanTableG[uint16_t(float(float(z / x) * 1024.0 + 0.5))];
    }

    return angle;
}

__device__ int16_t atan2sG(float z, float x) {
    int16_t angle = 0;

    if (x >= 0) {
        if (z >= 0) {
            if (z >= x) {
                angle = atan2_lookupG(x, z);
            }
            else {
                angle = 0x4000 - atan2_lookupG(z, x);
            }
        }
        else {
            z = -z;

            if (z < x) {
                angle = 0x4000 + atan2_lookupG(z, x);
            }
            else {
                angle = 0x8000 - atan2_lookupG(x, z);
            }
        }
    }
    else {
        x = -x;

        if (z < 0) {
            z = -z;

            if (z >= x) {
                angle = 0x8000 + atan2_lookupG(x, z);
            }
            else {
                angle = 0xC000 - atan2_lookupG(z, x);
            }
        }
        else {
            if (z < x) {
                angle = 0xC000 + atan2_lookupG(z, x);
            }
            else {
                angle = -atan2_lookupG(x, z);
            }
        }
    }

    return ((angle + 32768) % 65536) - 32768;
}

__device__ float find_closest_mag(float target) {
    int minIdx = -1;
    int maxIdx = magCount;

    while (maxIdx > minIdx + 1) {
        int midIdx = (maxIdx + minIdx) / 2;

        if (target < magSet[midIdx]) {
            maxIdx = midIdx;
        }
        else {
            minIdx = midIdx;
        }
    }

    if (minIdx == -1) {
        return magSet[maxIdx];
    }
    else if (maxIdx == magCount) {
        return magSet[minIdx];
    }
    else if (target - magSet[minIdx] < magSet[maxIdx] - target) {
        return magSet[minIdx];
    }
    else {
        return magSet[maxIdx];
    }
}

__global__ void init_mag_set() {
    bool magCheck[4097];

    for (int i = 0; i <= 4096; i++) {
        magCheck[i] = false;
    }

    for (int x = -128; x < 128; x++) {
        for (int y = -128; y < 128; y++) {
            int xS;
            if (x < 8) {
                if (x > -8) {
                    xS = 0;
                }
                else {
                    xS = x + 6;
                }
            }
            else {
                xS = x - 6;
            }
            int yS;
            if (y < 8) {
                if (y > -8) {
                    yS = 0;
                }
                else {
                    yS = y + 6;
                }
            }
            else {
                yS = y - 6;
            }

            int mag2 = xS * xS + yS * yS;
            mag2 = mag2 > 4096 ? 4096 : mag2;

            magCheck[mag2] = true;
        }
    }

    for (int i = 0; i <= 4096; i++) {
        if (magCheck[i]) {
            float mag = sqrtf((float)i);
            mag = (mag / 64.0f) * (mag / 64.0f) * 32.0f;
            magSet[magCount] = mag;
            magCount++;
        }
    }
}

__global__ void init_camera_angles() {
    for (int i = 0; i < 65536; i += 16) {
        int angle = atan2sG(gCosineTableG[i >> 4], gSineTableG[i >> 4]);
        angle = (65536 + angle) % 65536;

        validCameraAngle[angle] = true;
    }
}

__device__ int atan2b(double z, double x) {
    double A = 65536 * atan2(x, z) / (2 * M_PI);
    A = fmod(65536.0 + A, 65536.0);
    int lower = 0;
    int upper = 8192;

    while (upper - lower > 1) {
        int mid = (upper + lower) / 2;

        if (fmod(65536.0 + gArctanTableG[mid], 65536.0) > A) {
            upper = mid;
        }
        else {
            lower = mid;
        }
    }

    return lower;
}

__device__ int calculate_camera_yaw(float* currentPosition, float* lakituPosition) {
    short baseCameraYaw = -16384;
    float baseCameraDist = 1400.0;
    short baseCameraPitch = 0x05B0;
    short baseCameraFaceAngle = 24576;

    SurfaceG* floor;
    float floorY;

    float xOff = currentPosition[0] + gSineTableG[((65536 + (int)baseCameraYaw) % 65536) >> 4] * 40.f;
    float zOff = currentPosition[2] + gCosineTableG[((65536 + (int)baseCameraYaw) % 65536) >> 4] * 40.f;
    float offPos[3] = { xOff, currentPosition[1], zOff };

    int floorIdx = find_floor(offPos, &floor, floorY, floorsG, total_floorsG);
    floorY = floorY - currentPosition[1];

    if (floorIdx != -1) {
        if (floorY > 0) {
            if (!(floor->normal[2] == 0.f && floorY < 100.f)) {
                baseCameraPitch += atan2sG(40.f, floorY);
            }
        }
    }

    baseCameraPitch = baseCameraPitch + 2304;

    float cameraPos[3] = { currentPosition[0] + baseCameraDist * gCosineTableG[((65536 + (int)baseCameraPitch) % 65536) >> 4] * gSineTableG[((65536 + (int)baseCameraYaw) % 65536) >> 4],
                       currentPosition[1] + 125.0f + baseCameraDist * gSineTableG[((65536 + (int)baseCameraPitch) % 65536) >> 4],
                       currentPosition[2] + baseCameraDist * gCosineTableG[((65536 + (int)baseCameraPitch) % 65536) >> 4] * gCosineTableG[((65536 + (int)baseCameraYaw) % 65536) >> 4]
    };

    float pan[3] = { 0, 0, 0 };
    float temp[3] = { 0, 0, 0 };

    // Get distance and angle from camera to Mario.
    float dx = currentPosition[0] - cameraPos[0];
    float dy = currentPosition[1] + 125.0f;
    float dz = currentPosition[2] - cameraPos[2];

    float cameraDist = sqrtf(dx * dx + dy * dy + dz * dz);
    float cameraPitch = atan2sG(sqrtf(dx * dx + dz * dz), dy);
    float cameraYaw = atan2sG(dz, dx);

    // The camera will pan ahead up to about 30% of the camera's distance to Mario.
    pan[2] = gSineTableG[0xC0] * cameraDist;

    temp[0] = pan[0];
    temp[1] = pan[1];
    temp[2] = pan[2];

    pan[0] = temp[2] * gSineTableG[((65536 + (int)baseCameraFaceAngle) % 65536) >> 4] + temp[0] * gCosineTableG[((65536 + (int)baseCameraFaceAngle) % 65536) >> 4];
    pan[2] = temp[2] * gCosineTableG[((65536 + (int)baseCameraFaceAngle) % 65536) >> 4] + temp[0] * gSineTableG[((65536 + (int)baseCameraFaceAngle) % 65536) >> 4];

    // rotate in the opposite direction
    cameraYaw = -cameraYaw;

    temp[0] = pan[0];
    temp[1] = pan[1];
    temp[2] = pan[2];

    pan[0] = temp[2] * gSineTableG[((65536 + (int)cameraYaw) % 65536) >> 4] + temp[0] * gCosineTableG[((65536 + (int)cameraYaw) % 65536) >> 4];
    pan[2] = temp[2] * gCosineTableG[((65536 + (int)cameraYaw) % 65536) >> 4] + temp[0] * gSineTableG[((65536 + (int)cameraYaw) % 65536) >> 4];

    // Only pan left or right
    pan[2] = 0.f;

    cameraYaw = -cameraYaw;

    temp[0] = pan[0];
    temp[1] = pan[1];
    temp[2] = pan[2];

    pan[0] = temp[2] * gSineTableG[((65536 + (int)cameraYaw) % 65536) >> 4] + temp[0] * gCosineTableG[((65536 + (int)cameraYaw) % 65536) >> 4];
    pan[2] = temp[2] * gCosineTableG[((65536 + (int)cameraYaw) % 65536) >> 4] + temp[0] * gSineTableG[((65536 + (int)cameraYaw) % 65536) >> 4];

    float cameraFocus[3] = { currentPosition[0] + pan[0], currentPosition[1] + 125.0f + pan[1], currentPosition[2] + pan[2] };

    dx = cameraFocus[0] - lakituPosition[0];
    dy = cameraFocus[1] - lakituPosition[1];
    dz = cameraFocus[2] - lakituPosition[2];

    cameraDist = sqrtf(dx * dx + dy * dy + dz * dz);
    cameraPitch = atan2sG(sqrtf(dx * dx + dz * dz), dy);
    cameraYaw = atan2sG(dz, dx);

    if (cameraPitch > 15872) {
        cameraPitch = 15872;
    }
    if (cameraPitch < -15872) {
        cameraPitch = -15872;
    }

    cameraFocus[0] = lakituPosition[0] + cameraDist * gCosineTableG[((65536 + (int)cameraPitch) % 65536) >> 4] * gSineTableG[((65536 + (int)cameraYaw) % 65536) >> 4];
    cameraFocus[1] = lakituPosition[1] + cameraDist * gSineTableG[((65536 + (int)cameraPitch) % 65536) >> 4];
    cameraFocus[2] = lakituPosition[2] + cameraDist * gCosineTableG[((65536 + (int)cameraPitch) % 65536) >> 4] * gCosineTableG[((65536 + (int)cameraYaw) % 65536) >> 4];

    return atan2sG(lakituPosition[2] - cameraFocus[2], lakituPosition[0] - cameraFocus[0]);
}

__device__ void platform_logic_gpu(float* platform_normal, float* mario_pos, short(&triangles)[2][3][3], float(&normals)[2][3], float(&mat)[4][4]) {
    float dx;
    float dy;
    float dz;
    float d;

    float dist[3];
    float posBeforeRotation[3];
    float posAfterRotation[3];

    // Mario's position
    float mx = mario_pos[0];
    float my = mario_pos[1];
    float mz = mario_pos[2];

    dist[0] = mx - (float)platform_pos[0];
    dist[1] = my - (float)platform_pos[1];
    dist[2] = mz - (float)platform_pos[2];

    mat[1][0] = platform_normal[0];
    mat[1][1] = platform_normal[1];
    mat[1][2] = platform_normal[2];

    float invsqrt = 1.0f / sqrtf(mat[1][0] * mat[1][0] + mat[1][1] * mat[1][1] + mat[1][2] * mat[1][2]);

    mat[1][0] *= invsqrt;
    mat[1][1] *= invsqrt;
    mat[1][2] *= invsqrt;

    mat[0][0] = mat[1][1] * 1.0f - 0.0f * mat[1][2];
    mat[0][1] = mat[1][2] * 0.0f - 1.0f * mat[1][0];
    mat[0][2] = mat[1][0] * 0.0f - 0.0f * mat[1][1];

    invsqrt = 1.0f / sqrtf(mat[0][0] * mat[0][0] + mat[0][1] * mat[0][1] + mat[0][2] * mat[0][2]);

    mat[0][0] *= invsqrt;
    mat[0][1] *= invsqrt;
    mat[0][2] *= invsqrt;

    mat[2][0] = mat[0][1] * mat[1][2] - mat[1][1] * mat[0][2];
    mat[2][1] = mat[0][2] * mat[1][0] - mat[1][2] * mat[0][0];
    mat[2][2] = mat[0][0] * mat[1][1] - mat[1][0] * mat[0][1];

    invsqrt = 1.0f / sqrtf(mat[2][0] * mat[2][0] + mat[2][1] * mat[2][1] + mat[2][2] * mat[2][2]);

    mat[2][0] *= invsqrt;
    mat[2][1] *= invsqrt;
    mat[2][2] *= invsqrt;

    mat[3][0] = platform_pos[0];
    mat[3][1] = platform_pos[1];
    mat[3][2] = platform_pos[2];
    mat[0][3] = 0.0f;
    mat[1][3] = 0.0f;
    mat[2][3] = 0.0f;
    mat[3][3] = 1.0f;

    for (int i = 0; i < 3; i++) {
        posBeforeRotation[i] = mat[0][i] * dist[0] + mat[1][i] * dist[1] + mat[2][i] * dist[2];
    }

    dx = mx - (float)platform_pos[0];
    dy = 500.0f;
    dz = mz - (float)platform_pos[2];
    d = sqrtf(dx * dx + dy * dy + dz * dz);

    // Normalizing
    d = 1.0 / d;
    dx *= d;
    dy *= d;
    dz *= d;

    // Approach the normals by 0.01f towards the new goal, then create a transform matrix and orient the object. 
    // Outside of the other conditionals since it needs to tilt regardless of whether Mario is on.
    platform_normal[0] = (platform_normal[0] <= dx) ? ((dx - platform_normal[0] < 0.01f) ? dx : (platform_normal[0] + 0.01f)) : ((dx - platform_normal[0] > -0.01f) ? dx : (platform_normal[0] - 0.01f));
    platform_normal[1] = (platform_normal[1] <= dy) ? ((dy - platform_normal[1] < 0.01f) ? dy : (platform_normal[1] + 0.01f)) : ((dy - platform_normal[1] > -0.01f) ? dy : (platform_normal[1] - 0.01f));
    platform_normal[2] = (platform_normal[2] <= dz) ? ((dz - platform_normal[2] < 0.01f) ? dz : (platform_normal[2] + 0.01f)) : ((dz - platform_normal[2] > -0.01f) ? dz : (platform_normal[2] - 0.01f));

    mat[1][0] = platform_normal[0];
    mat[1][1] = platform_normal[1];
    mat[1][2] = platform_normal[2];

    invsqrt = 1.0f / sqrtf(mat[1][0] * mat[1][0] + mat[1][1] * mat[1][1] + mat[1][2] * mat[1][2]);

    mat[1][0] *= invsqrt;
    mat[1][1] *= invsqrt;
    mat[1][2] *= invsqrt;

    mat[0][0] = mat[1][1] * 1.0f - 0.0f * mat[1][2];
    mat[0][1] = mat[1][2] * 0.0f - 1.0f * mat[1][0];
    mat[0][2] = mat[1][0] * 0.0f - 0.0f * mat[1][1];

    invsqrt = 1.0f / sqrtf(mat[0][0] * mat[0][0] + mat[0][1] * mat[0][1] + mat[0][2] * mat[0][2]);

    mat[0][0] *= invsqrt;
    mat[0][1] *= invsqrt;
    mat[0][2] *= invsqrt;

    mat[2][0] = mat[0][1] * mat[1][2] - mat[1][1] * mat[0][2];
    mat[2][1] = mat[0][2] * mat[1][0] - mat[1][2] * mat[0][0];
    mat[2][2] = mat[0][0] * mat[1][1] - mat[1][0] * mat[0][1];

    invsqrt = 1.0f / sqrtf(mat[2][0] * mat[2][0] + mat[2][1] * mat[2][1] + mat[2][2] * mat[2][2]);

    mat[2][0] *= invsqrt;
    mat[2][1] *= invsqrt;
    mat[2][2] *= invsqrt;

    mat[3][0] = platform_pos[0];
    mat[3][1] = platform_pos[1];
    mat[3][2] = platform_pos[2];
    mat[0][3] = 0.0f;
    mat[1][3] = 0.0f;
    mat[2][3] = 0.0f;
    mat[3][3] = 1.0f;

    for (int i = 0; i < 3; i++) {
        posAfterRotation[i] = mat[0][i] * dist[0] + mat[1][i] * dist[1] + mat[2][i] * dist[2];
    }

    mx += posAfterRotation[0] - posBeforeRotation[0];
    my += posAfterRotation[1] - posBeforeRotation[1];
    mz += posAfterRotation[2] - posBeforeRotation[2];
    mario_pos[0] = mx;
    mario_pos[1] = my;
    mario_pos[2] = mz;
}